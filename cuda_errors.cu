#if defined(MPI_HDR)
extern "C" void clover_abort_();
#endif

#include "cuda_common.hpp"
#include <cstdarg>
#include <cstdio>

static const char* errorCodes
(int err_code)
{
    switch(err_code)
    {
        case hipSuccess: return "hipSuccess"; // 0
        case hipErrorMissingConfiguration: return "hipErrorMissingConfiguration"; // 1
        case hipErrorOutOfMemory: return "hipErrorOutOfMemory"; // 2
        case hipErrorNotInitialized: return "hipErrorNotInitialized"; // 3
        case hipErrorLaunchFailure: return "hipErrorLaunchFailure"; // 4
        case hipErrorPriorLaunchFailure: return "hipErrorPriorLaunchFailure"; // 5
        case hipErrorLaunchTimeOut: return "hipErrorLaunchTimeOut"; // 6
        case hipErrorLaunchOutOfResources: return "hipErrorLaunchOutOfResources"; // 7
        case hipErrorInvalidDeviceFunction: return "hipErrorInvalidDeviceFunction"; // 8
        case hipErrorInvalidConfiguration: return "hipErrorInvalidConfiguration"; // 9
        case hipErrorInvalidDevice: return "hipErrorInvalidDevice"; // 10
        case hipErrorInvalidValue: return "hipErrorInvalidValue";// 11
        case hipErrorInvalidPitchValue: return "hipErrorInvalidPitchValue";// 12
        case hipErrorInvalidSymbol: return "hipErrorInvalidSymbol";// 13
        case hipErrorMapFailed: return "hipErrorMapFailed";// 14
        case hipErrorUnmapFailed: return "hipErrorUnmapFailed";// 15
        // case cudaErrorInvalidHostPointer: return "cudaErrorInvalidHostPointer";// 16
        case hipErrorInvalidDevicePointer: return "hipErrorInvalidDevicePointer";// 17
        // case hipErrorInvalidTexture: return "hipErrorInvalidTexture";// 18
        // case cudaErrorInvalidTextureBinding: return "cudaErrorInvalidTextureBinding";// 19
        // case hipErrorInvalidChannelDescriptor: return "hipErrorInvalidChannelDescriptor";// 20
        case hipErrorInvalidMemcpyDirection: return "hipErrorInvalidMemcpyDirection";// 21
        // case cudaErrorAddressOfConstant: return "cudaErrorAddressOfConstant";// 22
        // case cudaErrorTextureFetchFailed: return "cudaErrorTextureFetchFailed";// 23
        // case cudaErrorTextureNotBound: return "cudaErrorTextureNotBound";// 24
        // case cudaErrorSynchronizationError: return "cudaErrorSynchronizationError";// 25
        // case cudaErrorInvalidFilterSetting: return "cudaErrorInvalidFilterSetting";// 26
        // case cudaErrorInvalidNormSetting: return "cudaErrorInvalidNormSetting";// 27
        // case cudaErrorMixedDeviceExecution: return "cudaErrorMixedDeviceExecution";// 28
        case hipErrorDeinitialized: return "hipErrorDeinitialized";// 29
        case hipErrorUnknown: return "hipErrorUnknown";// 30
        // case cudaErrorNotYetImplemented: return "cudaErrorNotYetImplemented";// 31
        // case cudaErrorMemoryValueTooLarge: return "cudaErrorMemoryValueTooLarge";// 32
        case hipErrorInvalidHandle: return "hipErrorInvalidHandle";// 33
        case hipErrorNotReady: return "hipErrorNotReady";// 34
        case hipErrorInsufficientDriver: return "hipErrorInsufficientDriver";// 35
        case hipErrorSetOnActiveProcess: return "hipErrorSetOnActiveProcess";// 36
        // case cudaErrorInvalidSurface: return "cudaErrorInvalidSurface";// 37
        case hipErrorNoDevice: return "hipErrorNoDevice";// 38
        case hipErrorECCNotCorrectable: return "hipErrorECCNotCorrectable";// 39
        case hipErrorSharedObjectSymbolNotFound: return "hipErrorSharedObjectSymbolNotFound";// 40
        case hipErrorSharedObjectInitFailed: return "hipErrorSharedObjectInitFailed";// 41
        case hipErrorUnsupportedLimit: return "hipErrorUnsupportedLimit";// 42
        // case cudaErrorDuplicateVariableName: return "cudaErrorDuplicateVariableName";// 43
        // case cudaErrorDuplicateTextureName: return "cudaErrorDuplicateTextureName";// 44
        // case cudaErrorDuplicateSurfaceName: return "cudaErrorDuplicateSurfaceName";// 45
        // case cudaErrorDevicesUnavailable: return "cudaErrorDevicesUnavailable";// 46
        case hipErrorInvalidImage: return "hipErrorInvalidImage";// 47
        case hipErrorNoBinaryForGpu: return "hipErrorNoBinaryForGpu";// 48
        // case cudaErrorIncompatibleDriverContext: return "cudaErrorIncompatibleDriverContext";// 49
        case hipErrorPeerAccessAlreadyEnabled: return "hipErrorPeerAccessAlreadyEnabled";// 50
        case hipErrorPeerAccessNotEnabled: return "hipErrorPeerAccessNotEnabled";// 51
        case hipErrorContextAlreadyInUse: return "hipErrorContextAlreadyInUse";// 52
        case hipErrorProfilerDisabled: return "hipErrorProfilerDisabled";// 53
        case hipErrorProfilerNotInitialized: return "hipErrorProfilerNotInitialized";// 54
        case hipErrorProfilerAlreadyStarted: return "hipErrorProfilerAlreadyStarted";// 55
        case hipErrorProfilerAlreadyStopped: return "hipErrorProfilerAlreadyStopped";// 56
        case hipErrorAssert: return "hipErrorAssert";// 57
        // case cudaErrorTooManyPeers: return "cudaErrorTooManyPeers";// 58
        case hipErrorHostMemoryAlreadyRegistered: return "hipErrorHostMemoryAlreadyRegistered";// 59
        case hipErrorHostMemoryNotRegistered: return "hipErrorHostMemoryNotRegistered";// 60
        case hipErrorOperatingSystem: return "hipErrorOperatingSystem";// 61
        // case cudaErrorStartupFailure: return "cudaErrorStartupFailure";// 62
        // case cudaErrorApiFailureBase: return "cudaErrorApiFailureBase";// 63
        default: return "Unknown error";
    }
}

void CloverleafCudaChunk::errorHandler
(int line_num, const char* file)
{
    hipDeviceSynchronize();
    int l_e = hipGetLastError();
    if (hipSuccess != l_e)
    {
        cloverDie(line_num, file, "Error in %s - return code %d (%s)\n", file, l_e, errorCodes(l_e));
    }
}

// print out timing info when done
CloverleafCudaChunk::~CloverleafCudaChunk
(void)
{
    if (profiler_on)
    {
        fprintf(stdout, "@@@@@ PROFILING @@@@@\n");

        for (std::map<std::string, double>::iterator ii = kernel_times.begin();
            ii != kernel_times.end(); ii++)
        {
            fprintf(stdout, "%35s : %.3f\n", ii->first.c_str(), ii->second);
        }
    }
}

std::vector<double> CloverleafCudaChunk::dumpArray
(const std::string& arr_name, int x_extra, int y_extra)
{
    std::vector<double> host_arr(BUFSZ2D(x_extra, y_extra)/sizeof(double));

    hipDeviceSynchronize();

    try
    {
        hipMemcpy(&host_arr.front(), arr_names.at(arr_name),
            BUFSZ2D(x_extra, y_extra), hipMemcpyDeviceToHost);
    }
    catch (std::out_of_range e)
    {
        DIE("Error - %s was not in the arr_names map\n", arr_name.c_str());
    }

    errorHandler(__LINE__, __FILE__);

    return host_arr;
}

// called when something goes wrong
void CloverleafCudaChunk::cloverDie
(int line, const char* filename, const char* format, ...)
{
    fprintf(stderr, "@@@@@\n");
    fprintf(stderr, "\x1b[31m");
    fprintf(stderr, "Fatal error at line %d in %s:", line, filename);
    fprintf(stderr, "\x1b[0m");
    fprintf(stderr, "\n");

    va_list arglist;
    va_start(arglist, format);
    vfprintf(stderr, format, arglist);
    va_end(arglist);

    // TODO add logging or something

    fprintf(stderr, "\nExiting\n");

#if defined(MPI_HDR)
    clover_abort_();
#else
    exit(1);
#endif
}

